#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "topk_impl.h"
#include "core/providers/cuda/cu_inc/common.cuh"
#include "hipcub/hipcub.hpp"
#include <limits>

namespace onnxruntime {
namespace cuda {

template <typename T>
__global__ void FillInput(const T* input_x, T* output_v, int64_t* output_i, const int64_t* elem_nums, size_t size, int64_t axis, int64_t K, int64_t offset, int64_t dimension) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, dimension);
  auto left = offset / (axis == size - 1 ? 1 : elem_nums[axis + 1]) * elem_nums[axis];
  auto right = axis == size - 1 ? 0 : offset % elem_nums[axis + 1];
  auto input_offset = left + id * (axis == size - 1 ? 1 : elem_nums[axis + 1]) + right;
  output_v[id] = input_x[input_offset];
  output_i[id] = id;
}

template <typename T>
__global__ void FillOutput(const T* input_v, const int64_t* input_i, T* output_v, int64_t* output_i, const int64_t* elem_nums, size_t size, int64_t axis, int64_t K, int64_t offset, int64_t dimension) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, dimension);
  auto left = offset / (axis == size - 1 ? 1 : elem_nums[axis + 1]) * elem_nums[axis] * K / dimension;
  auto right = axis == size - 1 ? 0 : offset % elem_nums[axis + 1];
  auto output_offset = left + id * (axis == size - 1 ? 1 : elem_nums[axis + 1]) + right;
  output_v[output_offset] = input_v[id];
  output_i[output_offset] = input_i[id];
}

__global__ void ExcludeOutput(int64_t* output_i, int64_t K, int64_t dimension) {
  CALCULATE_ELEMENTWISE_INDEX_OR_EXIT(id, dimension);
  if (id >= K) {
    output_i[id] = dimension;
  }
}

template <typename T>
Status TopKImpl(const CudaKernel* kernel, const T* input_x, T* output_v, int64_t* output_i, const int64_t* elem_nums, size_t size, int64_t axis, int64_t K, int64_t largest, int64_t sorted, int64_t N, int64_t dimension) {
  auto input_key_buffer = kernel->GetScratchBuffer<T>(dimension);
  auto output_key_buffer = kernel->GetScratchBuffer<T>(dimension);
  auto input_value_buffer = kernel->GetScratchBuffer<int64_t>(dimension);
  auto output_value_buffer = kernel->GetScratchBuffer<int64_t>(dimension);
  auto input_key = input_key_buffer.get();
  auto output_key = output_key_buffer.get();
  auto input_value = input_value_buffer.get();
  auto output_value = output_value_buffer.get();
  size_t temp_bytes = 0;
  CUDA_RETURN_IF_ERROR(hipcub::DeviceRadixSort::SortPairs(nullptr, temp_bytes, input_key, output_key, input_value, output_value, dimension));
  auto temp_storage_buffer = kernel->GetScratchBuffer<char>(temp_bytes);
  auto temp_storage = temp_storage_buffer.get();
  auto blocksPerGridD = (int)(ceil(static_cast<float>(dimension) / GridDim::maxThreadsPerBlock));
  auto blocksPerGridK = (int)(ceil(static_cast<float>(K) / GridDim::maxThreadsPerBlock));
  for (int64_t i = 0; i < N; i++) {
    FillInput<T><<<blocksPerGridD, GridDim::maxThreadsPerBlock, 0>>>(input_x, input_key, input_value, elem_nums, size, axis, K, i, dimension);
    CUDA_RETURN_IF_ERROR(1 == largest ? hipcub::DeviceRadixSort::SortPairsDescending(temp_storage, temp_bytes, input_key, output_key, input_value, output_value, dimension) : hipcub::DeviceRadixSort::SortPairs(temp_storage, temp_bytes, input_key, output_key, input_value, output_value, dimension));
    if (1 == sorted) {
      FillOutput<T><<<blocksPerGridK, GridDim::maxThreadsPerBlock, 0>>>(output_key, output_value, output_v, output_i, elem_nums, size, axis, K, i, dimension);
    } else {  //reorder by ascending index
      ExcludeOutput<<<blocksPerGridD, GridDim::maxThreadsPerBlock, 0>>>(output_value, K, dimension);
      CUDA_RETURN_IF_ERROR(hipcub::DeviceRadixSort::SortPairs(temp_storage, temp_bytes, output_value, input_value, output_key, input_key, dimension));
      FillOutput<T><<<blocksPerGridK, GridDim::maxThreadsPerBlock, 0>>>(input_key, input_value, output_v, output_i, elem_nums, size, axis, K, i, dimension);
    }
  }
  return Status::OK();
}

#define TOPKIMPLE(T) template Status TopKImpl<T>(const CudaKernel* kernel, \
                                                 const T* input_x,         \
                                                 T* output_v,              \
                                                 int64_t* output_i,        \
                                                 const int64_t* elem_nums, \
                                                 size_t size,              \
                                                 int64_t axis,             \
                                                 int64_t K,                \
                                                 int64_t largest,          \
                                                 int64_t sorted,           \
                                                 int64_t N,                \
                                                 int64_t dimension)

TOPKIMPLE(uint8_t);
TOPKIMPLE(uint16_t);
TOPKIMPLE(uint32_t);
TOPKIMPLE(uint64_t);
TOPKIMPLE(int8_t);
TOPKIMPLE(int16_t);
TOPKIMPLE(int32_t);
TOPKIMPLE(int64_t);
TOPKIMPLE(float);
TOPKIMPLE(double);

}  // namespace cuda
}  // namespace onnxruntime